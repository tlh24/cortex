#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <iostream>
#include <random>
#include "simsearch.h"

// this (w/ reduce) takes 4.2ms on 3080 laptop = slow.
__global__ void compute_distances_old(float* db, float* query, float* distances) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < DB_SIZE) {
        float distance = 0;
        for (int i = 0; i < DIM; i++) {
            float diff = db[idx * DIM + i] - query[i];
            distance += diff * diff;
        }
        distances[idx] = distance;
    }
}

__global__ void compute_distances2(unsigned char* db, unsigned char* query, float* dist)
{
	__shared__ float sacc[BLOCK_STRIDE][32];
	//__shared__ unsigned char qq[DIM]; // this makes no difference. 
	// hidden by the HW cache. 
	
	int x = threadIdx.x;
	int y = threadIdx.y;
	int by = blockIdx.x * BLOCK_STRIDE + y;

	float acc = 0.0;
	for( int i = 0; i < WARPSTEPS; i++ ){
		int j = i*32 + x;
		// this assumes everything past DIMSHORT is 0. 
		float diff = (float)(db[by*DIM + j] - query[j]);
		acc += diff * diff;
	}
	sacc[y][x] = acc;
	__syncthreads();
	// if(x < 16) sacc[y][x] += sacc[y][x + 16];
	// if(x < 8 ) sacc[y][x] += sacc[y][x + 8 ];
	// if(x < 4 ) sacc[y][x] += sacc[y][x + 4 ];
	// if(x < 2 ) sacc[y][x] += sacc[y][x + 2 ];
	// if(x < 1 ) sacc[y][x] += sacc[y][x + 1 ];
 // 
	// if(x == 0) dist[by] = sacc[y][x];
	
	if(x < 16) { 
		sacc[y][x] += sacc[y][x + 16];
		__syncthreads(); // why is this needed ??? 
		sacc[y][x] += sacc[y][x + 8 ];
		__syncthreads(); // threads in a warp should be synchronous.
		sacc[y][x] += sacc[y][x + 4 ];
		__syncthreads();
		sacc[y][x] += sacc[y][x + 2 ];
		__syncthreads();
		sacc[y][x] += sacc[y][x + 1 ];
		__syncthreads();
		if(x == 0) dist[by] = sacc[y][x];
	}
}

__device__ void warpReduce(volatile float *minDist, 
									volatile int *minIndx, unsigned int tid) {
	// all threads in a warp are synchronous
	// so you don't need to call syncthreads
	// and don't need if(tid < stride)
	for( int stride = 32; stride > 0; stride /= 2){
		if(minDist[tid] > minDist[tid + stride]){
			minDist[tid] = minDist[tid + stride]; 
			minIndx[tid] = minIndx[tid + stride]; 
		}
	}
}  

__global__ void findMinOfArray(float *dists, float *outDist, int* outIndx)
{
	unsigned int row = blockIdx.x * blockDim.x + threadIdx.x; /* unique id for each thread in the block*/

	unsigned int thread_id = threadIdx.x; /* thread index in the block*/

	__shared__ float minDist[BLOCK_SIZE];
	__shared__ int minIndx[BLOCK_SIZE];

	// load local data. Don't need to check dims, by construction.
	minDist[thread_id] = dists[row];
	minIndx[thread_id] = row; 
	__syncthreads();

	for(unsigned int stride = (blockDim.x/2); stride > 32 ; stride /=2){
		if(thread_id < stride){
			if(minDist[thread_id] > minDist[thread_id + stride]){ 
				minDist[thread_id] = minDist[thread_id + stride]; 
				minIndx[thread_id] = minIndx[thread_id + stride]; 
			}
		}
		__syncthreads();
	}

	if(thread_id < 32){
		warpReduce(minDist, minIndx, thread_id);
	}

	if(thread_id == 0){
		outDist[blockIdx.x] = minDist[0];
		outIndx[blockIdx.x] = minIndx[0];
	}
}

imgdb* simdb_allocate(int num)
{
	if(num != DB_SIZE){
		printf("simdb_allocate: asked for %d, compiled with %d\n",
			num, DB_SIZE);
		return 0;
	}
	imgdb* sdb = new imgdb;

	hipMalloc(&(sdb->db), DB_SIZE * DIM * sizeof(unsigned char));
	hipMalloc(&(sdb->query), DIM * sizeof(unsigned char));
	hipMalloc(&(sdb->distances), DB_SIZE * sizeof(float));
	hipMalloc(&(sdb->outDist), NUM_BLOCKS * sizeof(float));
	hipMalloc(&(sdb->outIndx), NUM_BLOCKS * sizeof(int));
	
	hipMemset(sdb->db, 0, DB_SIZE * DIM);
	hipMemset(sdb->query, 0, DIM); // never copy past DIMSHORT

	sdb->h_outDist = (float*)malloc(NUM_BLOCKS*sizeof(float));
	sdb->h_outIndx = (int*)malloc(NUM_BLOCKS*sizeof(int));

	return sdb;
}

void simdb_free(imgdb* sdb)
{
	hipFree(sdb->db);
	hipFree(sdb->query);
	hipFree(sdb->distances);
	hipFree(sdb->outDist);
	hipFree(sdb->outIndx);

	free(sdb->h_outDist);
	free(sdb->h_outIndx);
	free(sdb);
}

void simdb_set(imgdb* sdb, int i, unsigned char* row)
{
	if( i>=0 && i < DB_SIZE)
		hipMemcpy(sdb->db + i*DIM, row, DIMSHORT,
				  hipMemcpyHostToDevice);
}

void simdb_get(imgdb* sdb, int i, unsigned char* row)
{
	if( i>=0 && i < DB_SIZE)
		hipMemcpy(row, sdb->db + i*DIM, DIMSHORT,
				  hipMemcpyDeviceToHost);
}

void simdb_query(imgdb* sdb, unsigned char* query,
				float* minDist, int* minIndx)
{
	// check if query is a blank image (all ones or all zeros)
	// do it on the CPU: simpler / faster.  GCC should auto-vectorize.
	int sum = 0; 
	for(int i=0; i<DIMSHORT; i++){
		sum += query[i]; 
	}
	if(sum == DIMSHORT*255 || sum == 0){
		*minDist = -1.0; 
		*minIndx = 0; 
	} else {
		hipMemcpy(sdb->query, query, DIMSHORT,
					hipMemcpyHostToDevice);

		dim3 dimBlock(32, BLOCK_STRIDE, 1); // x, y, z
		
		// each warp has 32 threads, and computes the MSE of one row. 
		// each block has 256 threads, so computes thet MSE of 8 rows. 
		// hence, BLOCK_STRIDE = 8
		// there are DB_SIZE / BLOCK_STRIDE blocks. 

		compute_distances2<<<DB_SIZE/BLOCK_STRIDE, dimBlock>>>
				(sdb->db, sdb->query, sdb->distances);

		findMinOfArray<<<NUM_BLOCKS, BLOCK_SIZE>>>
				(sdb->distances, sdb->outDist, sdb->outIndx);

		hipMemcpy(sdb->h_outDist, sdb->outDist, sizeof(float)*NUM_BLOCKS, hipMemcpyDeviceToHost);
		hipMemcpy(sdb->h_outIndx, sdb->outIndx, sizeof(int)*NUM_BLOCKS, hipMemcpyDeviceToHost);

		float d = sdb->h_outDist[0];
		int n = sdb->h_outIndx[0];
		for(int i=1; i<NUM_BLOCKS; i++){
			if(sdb->h_outDist[i] < d){
				d = sdb->h_outDist[i];
				n = sdb->h_outIndx[i];
			}
		}
		*minDist = d;
		*minIndx = n;
		
		// checksies ~
		// float* m = (float*)malloc(sizeof(float)*DB_SIZE); 
		// hipMemcpy(m, sdb->distances, sizeof(float)*DB_SIZE,
		// 			hipMemcpyDeviceToHost);
		// d = m[0]; 
		// n = 0; 
		// for( int i=1; i<DB_SIZE; i++ ){
		// 	if( m[i] < d ){
		// 		d = m[i]; 
		// 		n = i; 
		// 	}
		// }
		// if( *minDist != d || *minIndx != n ){
		// 	printf("findMinOfArray error! [%d] %f should be [%d] %f", 
		// 			*minIndx, *minDist, n, d); 
		// }
		// free(m); 
	}
}

double simdb_checksum(imgdb* sdb)
{
	//checksum the whole database. 
	unsigned char* buf = (unsigned char*)malloc(DB_SIZE * DIM); 
	hipMemcpy(buf, sdb->db, DB_SIZE * DIM,
				  hipMemcpyDeviceToHost);
	
	double sum = 0.0; 
	for(int j=0; j<DB_SIZE; j++){
		for(int i=0; i<DIMSHORT;i++){
			sum += buf[j*DIM + i] / 255.0; 
		}
	}
	
	free(buf); 
	return sum; 
}

void simdb_clear(imgdb* sdb)
{
	hipMemset(sdb->db, 0, DB_SIZE * DIM);
	hipMemset(sdb->query, 0, DIM);
}
